
#include <hip/hip_runtime.h>
#include <stdio.h>

#define ROW_A 2
#define COL_A 3
#define COL_B 2

// CUDA kernel for matrix multiplication
__global__ void matrixMul(int* a, int* b, int* c) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int sum = 0;
    for (int k = 0; k < COL_A; k++) {
        sum += a[row * COL_A + k] * b[k * COL_B + col];
    }

    c[row * COL_B + col] = sum;
}

int main() {
    int a[ROW_A][COL_A] = {{1, 2, 3}, {4, 5, 6}};
    int b[COL_A][COL_B] = {{7, 8}, {9, 10}, {11, 12}};
    int c[ROW_A][COL_B];  // Output matrix

    int *dev_a, *dev_b, *dev_c;  // Device copies of input and output matrices

    // Allocate memory on the device
    hipMalloc((void**)&dev_a, ROW_A * COL_A * sizeof(int));
    hipMalloc((void**)&dev_b, COL_A * COL_B * sizeof(int));
    hipMalloc((void**)&dev_c, ROW_A * COL_B * sizeof(int));

    // Copy input matrices from host to device
    hipMemcpy(dev_a, a, ROW_A * COL_A * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, COL_A * COL_B * sizeof(int), hipMemcpyHostToDevice);

    // Set grid and block dimensions
    dim3 blockDim(COL_B, ROW_A);
    dim3 gridDim(1, 1);

    // Launch kernel
    matrixMul<<<gridDim, blockDim>>>(dev_a, dev_b, dev_c);

    // Copy output matrix from device to host
    hipMemcpy(c, dev_c, ROW_A * COL_B * sizeof(int), hipMemcpyDeviceToHost);

    // Print the output matrix
    for (int i = 0; i < ROW_A; i++) {
        for (int j = 0; j < COL_B; j++) {
            printf("%d\t", c[i][j]);
        }
        printf("\n");
    }

    // Free device memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
